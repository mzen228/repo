#include "hip/hip_runtime.h"
﻿/*Add two arrays
Confirm GPU results with CPU results*/
#include "hip/hip_runtime.h"
#include ""

#include "cuda_common.h"
#include <stdio.h>

// for random initialize
#include <stdlib.h>
#include <time.h>

// for memset
#include <cstring>

__global__ void  sum_array_gpu(int* a, int* b, int* c, int size) {
	int gid = blockIdx.x * blockDim.x + threadIdx.x;

	if (gid < size) {
		c[gid] = a[gid] + b[gid];
	}
}

void sum_array_cpu(int* a, int* b, int* c, int size) {
	for (int i{}; i < size; ++i) {
		c[i] = a[i] + b[i];
	}
}

// compare two arrays
void compare_arrays(int* a, int* b, int size) {
	for (int i{}; i < size; ++i) {
		if (a[i] != b[i]) {
			printf("Arrays are not equal\n");
			return;
		}
	}

	printf("Arrays are equal\n");
	return;
}

int main() {
	helloWorld();
	int size = 10000;
	int block_size = 128;

	int NO_BYTES = size * sizeof(int);

	// host pointers
	int* h_a, * h_b, * gpu_results, * h_c;

	// allocate memory for host pointers 
	h_a = (int*)malloc(NO_BYTES);
	h_b = (int*)malloc(NO_BYTES);
	gpu_results = (int*)malloc(NO_BYTES);
	h_c = (int*)malloc(NO_BYTES);

	// initialize host pointer
	time_t t;
	srand((unsigned)time(&t));

	for (int i{}; i < size; ++i) {
		h_a[i] = (int)(rand() & 0xFF); // limit random numbers to range of 0 to 255
	}

	for (int i{}; i < size; ++i) {
		h_b[i] = (int)(rand() & 0xFF);
	}

	sum_array_cpu(h_a, h_b, h_c, size);

	memset(gpu_results, 0, NO_BYTES);

	// device pointer
	int* d_a, * d_b, * d_c;
	gpuErrchk(hipMalloc((int**)&d_a, NO_BYTES));
	gpuErrchk(hipMalloc((int**)&d_b, NO_BYTES));
	gpuErrchk(hipMalloc((int**)&d_c, NO_BYTES));

	// memory transfer from host to device 
	hipMemcpy(d_a, h_a, NO_BYTES, hipMemcpyHostToDevice);
	hipMemcpy(d_b, h_b, NO_BYTES, hipMemcpyHostToDevice);

	// launching the grid
	dim3 block(block_size);
	dim3 grid((size / block.x) + 1);

	sum_array_gpu << <grid, block >> > (d_a, d_b, d_c, size);
	hipDeviceSynchronize();

	// memory transfer back to host 
	hipMemcpy(gpu_results, d_c, NO_BYTES, hipMemcpyDeviceToHost);

	// array comparison 
	compare_arrays(gpu_results, h_c, size);

	hipFree(d_c);
	hipFree(d_b);
	hipFree(d_a);

	free(gpu_results);
	free(h_b);
	free(h_a);

	hipDeviceReset();

	return 0;


}