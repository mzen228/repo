#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>

__global__ void unique_idx_cal_threadIdx(int * input) {
	int tid = threadIdx.x;
	int gid = tid + blockIdx.x * blockDim.x; 
	printf("gid: %d, value: %d\n", gid, input[gid]);
}

int main() {
	int array_size = 16;
	int array_byte_size = array_size * sizeof(int);
	int h_data[] = { 23,9,4,54,65,12,1,33,52, 106, 48, 26, 18, 91, 85, 75 }; 

	printf("Array: ");
	for (int i{}; i < array_size; ++i)
		printf("%d, ", h_data[i]);
	printf("\n\n");

	int* d_data;
	hipMalloc((void**)&d_data, array_byte_size);
	hipMemcpy(d_data, h_data, array_byte_size, hipMemcpyHostToDevice);

	dim3 block(4);
	dim3 grid(4); 

	unique_idx_cal_threadIdx << <grid, block >> > (d_data);
	hipDeviceSynchronize();
	hipDeviceReset();
}