#include "hip/hip_runtime.h"
﻿/*Add two arrays
Confirm GPU results with CPU results*/
#include "hip/hip_runtime.h"
#include ""

#include "../SumArray/cuda_common.h"
#include <stdio.h>

// for random initialize
#include <stdlib.h>
#include <time.h>

// for memset
#include <cstring>

int main() {
	//int size = 10000;
	//int block_size = 128;
	int size = 1 << 25;
	int block_size = 1024;

	int NO_BYTES = size * sizeof(int);

	// host pointers
	int* h_a, * h_b, * gpu_results, * h_c;

	// allocate memory for host pointers 
	h_a = (int*)malloc(NO_BYTES);
	h_b = (int*)malloc(NO_BYTES);
	gpu_results = (int*)malloc(NO_BYTES);
	h_c = (int*)malloc(NO_BYTES);

	// initialize host pointer
	time_t t;
	srand((unsigned)time(&t));

	for (int i{}; i < size; ++i) {
		h_a[i] = (int)(rand() & 0xFF); // limit random numbers to range of 0 to 255
	}

	for (int i{}; i < size; ++i) {
		h_b[i] = (int)(rand() & 0xFF);
	}

	// summation in CPU
	clock_t cpu_start, cpu_end, gpu_start, gpu_end, htod_start, htod_end,
		dtoh_start, dtoh_end;

	cpu_start = clock();
	sum_array_cpu(h_a, h_b, h_c, size);
	cpu_end = clock();

	memset(gpu_results, 0, NO_BYTES);

	// device pointer
	int* d_a, * d_b, * d_c;
	hipMalloc((int**)&d_a, NO_BYTES);
	hipMalloc((int**)&d_b, NO_BYTES);
	hipMalloc((int**)&d_c, NO_BYTES);

	// memory transfer from host to device 
	htod_start = clock();
	hipMemcpy(d_a, h_a, NO_BYTES, hipMemcpyHostToDevice);
	hipMemcpy(d_b, h_b, NO_BYTES, hipMemcpyHostToDevice);
	htod_end = clock();

	// launching the grid
	dim3 block(block_size);
	dim3 grid((size / block.x) + 1);

	gpu_start = clock();
	sum_array_gpu << <grid, block >> > (d_a, d_b, d_c, size);
	gpu_end = clock(); 
	hipDeviceSynchronize();

	// memory transfer back to host 
	dtoh_start = clock();
	hipMemcpy(gpu_results, d_c, NO_BYTES, hipMemcpyDeviceToHost);
	dtoh_end = clock();

	// array comparison 
	compare_arrays(gpu_results, h_c, size);

	printf("Sum array CPU execution time: %4.8f\n",
		(double)((double)(cpu_end - cpu_start) / CLOCKS_PER_SEC));

	printf("Sum array GPU execution time: %4.8f\n",
		(double)((double)(gpu_end - gpu_start) / CLOCKS_PER_SEC));

	printf("htod mem transfer time: %4.8f\n",
		(double)((double)(htod_end - htod_start) / CLOCKS_PER_SEC));

	printf("dtoh mem transfer time: %4.8f\n",
		(double)((double)(dtoh_end - dtoh_start) / CLOCKS_PER_SEC));

	printf("Total GPU execution time: %4.8f\n",
		(double)((double)(dtoh_end - htod_start) / CLOCKS_PER_SEC));

	hipFree(d_c);
	hipFree(d_b);
	hipFree(d_a);

	free(gpu_results);
	free(h_b);
	free(h_a);

	hipDeviceReset();

	return 0;


}